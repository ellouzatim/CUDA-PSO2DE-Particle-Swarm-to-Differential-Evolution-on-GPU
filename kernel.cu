#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "kernel.h"

__device__ float tempParticle[NUM_OF_DIMENSIONS];
__device__ float tempParticle2[NUM_OF_DIMENSIONS];

__device__ float tempParticleOld[NUM_OF_DIMENSIONS];
__device__ float tempParticleMutation[NUM_OF_DIMENSIONS];

/* Objective function
0: Levy 3-dimensional
1: Shifted Rastigrin's Function
2: Shifted Rosenbrock's Function
3: Shifted Griewank's Function
4: Shifted Sphere's Function
*/
/**
 * Runs on the GPU, called from the GPU.
*/

__device__ float device_fitness_function(float x[]) {
    float res = 0;
    float somme = 0;
    float produit = 0;

    switch (SELECTED_OBJ_FUNC) {
        case 0: {
            float y1 = 1 + (x[0] - 1) / 4;
            float yn = 1 + (x[NUM_OF_DIMENSIONS - 1] - 1) / 4;

            res += pow(sin(phi * y1), 2);

            for (int i = 0; i < NUM_OF_DIMENSIONS - 1; i++) {
                float y = 1 + (x[i] - 1) / 4;
                float yp = 1 + (x[i + 1] - 1) / 4;
                res += pow(y - 1, 2) * (1 + 10 * pow(sin(phi * yp), 2)) + pow(yn - 1, 2);
            }
            break;
        }
        case 1: {
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2) - 10 * cos(2 * phi * zi) + 10;
            }
            res -= 330;
            break;
        }
        case 2:
            for (int i = 0; i < NUM_OF_DIMENSIONS - 1; i++) {
                float zi = x[i] - 0 + 1;
                float zip1 = x[i + 1] - 0 + 1;
                res += 100 * (pow(pow(zi, 2) - zip1, 2)) + pow(zi - 1, 2);
            }
            res += 390;
            break;
        case 3:
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                somme += pow(zi, 2) / 4000;
                produit *= cos(zi / pow(i + 1, 0.5));
            }
            res = somme - produit + 1 - 180;
            break;
        case 4:
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2);
            }
            res -= 450;
            break;
    }

    return res;
}

__global__ void kernelInitializePopulation(float *population, hiprandState *states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // avoid an out of bound for the array
    if (i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS)
        return;

    hiprandState localState = states[i / NUM_OF_DIMENSIONS];
    // random entre
    population[i] = START_RANGE_MIN + hiprand_uniform(&localState) * (START_RANGE_MAX - START_RANGE_MIN);
}

__global__ void kernelEvaluerPopulationInitiale(float *population, float *evaluation) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0) // /!\i >= NUM_OF_POPULATION  est trop petit
        return;

    float tempParticle[NUM_OF_DIMENSIONS];
    for (int j = 0; j < NUM_OF_DIMENSIONS; j++) {
        tempParticle[j] = population[i + j];
    }
    evaluation[i / NUM_OF_DIMENSIONS] = device_fitness_function(tempParticle);
}

/**
 * Initialize a hiprandState
 *
 * @param states Array to store hiprandState objects
 * @param seed Seed for random number generation
 */

__global__ void setupCurand(hiprandState *states, unsigned long long seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < NUM_OF_POPULATION || i % NUM_OF_DIMENSIONS != 0) {
        hiprand_init(seed, i, 0, &states[i]);
    }
}

/**
 * Randomly create an array of indices for mutation.
 * For each individual, generate 3 distinct indices that are different from i, within the range 0 to NUM_OF_POPULATION.
 *
 * @param indexMutation Integer array to store the indices
 * @param states State array for random generation
 */

__global__ void kernelPrepareMutation(int *indexMutation, hiprandState *states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= NUM_OF_POPULATION || i % NUM_OF_DIMENSIONS != 0)
        return;

    int offsetIndividu = i / NUM_OF_DIMENSIONS;
    int offsetIndexMutation = offsetIndividu * 3;

    hiprandState localState = states[offsetIndividu];
    int used[NUM_OF_POPULATION] = {0};

    int count = 0;
    int attempts = 0;
    while (count < 3 && attempts < NUM_OF_POPULATION * 2) {
        unsigned int randomIdx = hiprand(&localState) % NUM_OF_POPULATION;
        if (randomIdx != offsetIndividu && !used[randomIdx]) {
            indexMutation[offsetIndexMutation + count] = randomIdx;
            used[randomIdx] = 1;
            count++;
            attempts++;
        }
    }
}

__global__ void kernelDEMutation(float *individuals, int *indexMutation, float *mutants, float F) {
    extern __shared__ float sharedMem[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0) return;

    int r_base = indexMutation[i * 3];
    int r_1 = indexMutation[i * 3 + 1];
    int r_2 = indexMutation[i * 3 + 2];

    float *base = &sharedMem[threadIdx.x * NUM_OF_DIMENSIONS * 3];
    float *x_r1 = &base[NUM_OF_DIMENSIONS];
    float *x_r2 = &x_r1[NUM_OF_DIMENSIONS];

    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        base[d] = individuals[r_base * NUM_OF_DIMENSIONS + d];
        x_r1[d] = individuals[r_1 * NUM_OF_DIMENSIONS + d];
        x_r2[d] = individuals[r_2 * NUM_OF_DIMENSIONS + d];
    }

    __syncthreads();

    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        mutants[i * NUM_OF_DIMENSIONS + d] = base[d] + F * (x_r1[d] - x_r2[d]);
    }
}

/**
 * Crossover DE
 * Update the values of the param mutated_individuals with the crossover
 *
 * Params :
 *  - previous_individuals : current population
 *  - mutated_individuals : population with mutation
 *  - k : random [0, D-1], D = dimension (generated each iteartion)
 */
__global__ void kernelCrossoverDE(
    float *previous_individuals,
    float *mutated_individuals,
    int k,
    hiprandState *states
) {
    // id du processus
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // avoid an out of bound for the array
    if (i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS)
        return;

    // individual : ceil(i / NUM_OF_DIMENSIONS), not useful to compute here

    // j : current index the individual
    int j = i % NUM_OF_DIMENSIONS;
    hiprandState localState = states[i / NUM_OF_DIMENSIONS];
    float randj = hiprand_uniform(&localState);

    // cf. crossover, equation (2) in the paper
    if (!(randj <= CR || j == k)) {
        // <=> vector U(i,j) in the paper
        mutated_individuals[i] = previous_individuals[i];
    }
}

__global__ void kernelEvaluerPopulation(float *oldPopulation, float *mutatedPopulation) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // avoid an out of bound for the array
    if (i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++) {
        tempParticleOld[j] = oldPopulation[i + j];
        tempParticleMutation[j] = mutatedPopulation[i + j];
    }

    if (!(device_fitness_function(tempParticleOld) > device_fitness_function(tempParticleMutation))) {
        for (int j = 0; j < NUM_OF_DIMENSIONS; j++) {
            oldPopulation[i + j] = tempParticleMutation[j];
        }
    }
}

extern "C" void cuda_de(float *population, float *gBest) {
    int size = NUM_OF_POPULATION * NUM_OF_DIMENSIONS;

    float *devPopulation;
    float *devEval;
    //float *devGBest; //not used
    float *devMutants;
    int *devIndexMutation;
    float evaluation[NUM_OF_POPULATION];  // float evaluation[NUM_OF_POPULATION * NUM_OF_DIMENSIONS]; /!\incohérence avec devEval qui est seulement de taille NUM_OF_POPULATION
    hiprandState *dstatesInitPop;
    hiprandState *dstatesPrepareMutation;
    hiprandState *dstatesCrossover;


    hipMalloc((void**)&devPopulation, sizeof(float) * size);
    hipMalloc((void**)&devEval, sizeof(float) * NUM_OF_POPULATION);
    hipMalloc((void**)&devMutants, sizeof(float) * size);
    hipMalloc((void**)&devIndexMutation, sizeof(int) * NUM_OF_POPULATION * 3);
    hipMalloc((void**)&dstatesInitPop, sizeof(hiprandState) * NUM_OF_POPULATION);
    hipMalloc((void**)&dstatesPrepareMutation, sizeof(hiprandState) * NUM_OF_POPULATION);
    hipMalloc((void**)&dstatesCrossover, sizeof(hiprandState) * size);
    // hipMalloc((void**)&devGBest, sizeof(float) * NUM_OF_DIMENSIONS);

    int threadsNum = 256;
    int blocksNum = (NUM_OF_POPULATION + threadsNum - 1) / threadsNum;

    size_t sharedMemSize = threadsNum * NUM_OF_DIMENSIONS * 3 * sizeof(float);

    hipMemcpy(devPopulation, population, sizeof(float) * size, hipMemcpyHostToDevice);
    //hipMemcpy(devEval, evaluation, sizeof(float) * NUM_OF_POPULATION, hipMemcpyHostToDevice); // /!\copie de données non initialisées vers le gpu 


    // Initialisation
    setupCurand<<<blocksNum, threadsNum>>>(dstatesInitPop, time(NULL));
    kernelInitializePopulation<<<blocksNum, threadsNum>>>(devPopulation, dstatesInitPop);
    kernelEvaluerPopulationInitiale<<<blocksNum, threadsNum>>>(devPopulation, devEval); // /!\ ne sert à rien pour l'instant

    for (int i = 0; i < size; i += NUM_OF_DIMENSIONS)
        gBest[i] = population[i];


    for (int iter = 0; iter < MAX_ITER; iter++) {
        float *tempPopulation = new float[size];  //tableau temporaire pour toute la population
        float tempIndividual[NUM_OF_DIMENSIONS]; //pour un individu

        setupCurand<<<blocksNum, threadsNum>>>(dstatesPrepareMutation, time(NULL));
        
        kernelPrepareMutation<<<blocksNum, threadsNum>>>(devIndexMutation, dstatesPrepareMutation);
        
        kernelDEMutation<<<blocksNum, threadsNum, sharedMemSize>>>(devPopulation, devIndexMutation, devMutants, F);
        
        int r = getRandom(0, NUM_OF_DIMENSIONS - 1);
        setupCurand<<<blocksNum, threadsNum>>>(dstatesCrossover, time(NULL));
        kernelCrossoverDE<<<blocksNum, threadsNum>>>(devPopulation, devMutants, r, dstatesCrossover);

        // Ajoutez ici le kernel de sélection si nécessaire
        kernelEvaluerPopulation<<<blocksNum, threadsNum>>>(devPopulation, devMutants);
        
        hipMemcpy(tempPopulation, devPopulation, sizeof(float) * size, hipMemcpyDeviceToHost);
        
        // compute current global best
        for (int i = 0; i < size; i += NUM_OF_DIMENSIONS) {
            for (int j = 0; j < NUM_OF_DIMENSIONS; j++) {
                tempIndividual[j] = tempPopulation[i + j]; // nouvelle version qui utilise un tableau temporaire tempPopulation pour stocker devPopulation depuis le gpu car on ne peut pas accéder à devPopulation depuis le cpu
            }
            if (host_fitness_function(tempIndividual) < host_fitness_function(gBest)) {
                for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
                    gBest[k] = tempIndividual[k];
            }
        }
        delete[] tempPopulation;
    }

    hipMemcpy(population, devPopulation, sizeof(float) * size, hipMemcpyDeviceToHost);

    hipFree(devPopulation);
    hipFree(devEval);
    hipFree(devMutants);
    hipFree(devIndexMutation);
    hipFree(dstatesPrepareMutation);
    hipFree(dstatesCrossover);
    //hipFree(devGBest);
}

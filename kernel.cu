#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "kernel.h"




__device__ float tempParticle[NUM_OF_DIMENSIONS];
__device__ float tempParticle2[NUM_OF_DIMENSIONS];


/* Objective function
0: Levy 3-dimensional
1: Shifted Rastigrin's Function
2: Shifted Rosenbrock's Function
3: Shifted Griewank's Function
4: Shifted Sphere's Function
*/
/**
 * Runs on the GPU, called from the GPU.
*/


__global__ void kernelInitializePopulation(float *population)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // avoid an out of bound for the array
    if(i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS)
        return;
    population[i] = getRandom(START_RANGE_MIN,START_RANGE_MAX);
}

__global__ void kernelEvaluerPopulation(float*population,float *evaluation)
{    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // avoid an out of bound for the array
        if(i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle[j] = population[i + j];
    }
    evaluation[i] = fitness_function(tempParticle);
}


__device__ float fitness_function(float x[]) {
    float res = 0;
    float somme = 0;
    float produit = 0;


    switch (SELECTED_OBJ_FUNC)  {
        case 0:
            float y1 = 1 + (x[0] - 1)/4;
            float yn = 1 + (x[NUM_OF_DIMENSIONS-1] - 1)/4;


            res += pow(sin(phi*y1), 2);


            for (int i = 0; i < NUM_OF_DIMENSIONS-1; i++) {
                float y = 1 + (x[i] - 1)/4;
                float yp = 1 + (x[i+1] - 1)/4;
                res += pow(y - 1, 2)*(1 + 10*pow(sin(phi*yp), 2)) + pow(yn - 1, 2);
            }
            break;
        case 1:
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2) - 10*cos(2*phi*zi) + 10;
            }
            res -= 330;
            break;
       
        case 2:
            for (int i = 0; i < NUM_OF_DIMENSIONS-1; i++) {
                float zi = x[i] - 0 + 1;
                float zip1 = x[i+1] - 0 + 1;
                res += 100 * ( pow(pow(zi, 2) - zip1, 2)) + pow(zi - 1, 2);
            }
            res += 390;  
            break;
        case 3:
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                somme += pow(zi, 2)/4000;
                produit *= cos(zi/pow(i+1, 0.5));
            }
            res = somme - produit + 1 - 180;
            break;
        case 4:
            for(int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2);
            }
            res -= 450;
            break;
    }  

/**
 * Crossover DE
 * Update the values of the param mutated_individuals with the crossover
 *
 * Params :
 *  - previous_individuals : current population
 *  - mutated_individuals : population with mutation
 *  - k : random [0, D-1], D = dimension (generated each iteartion) 
*/
__global__ void kernelCrossoverDE (
    float *previous_individuals, 
    float *mutated_individuals, 
    int k
    )
{
    // id du processus
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // avoid an out of bound for the array 
    if(i >= NUM_OF_POPULATION * NUM_OF_DIMENSIONS)
        return;

    // individual : ceil(i / NUM_OF_DIMENSIONS), not useful to compute here

    // j : current index the individual 
    int j = i % NUM_OF_DIMENSIONS; 
    float randj = getRandomClamped(); // random [0, 1], cf. kernel.cpp
    
    // cf. crossover, equation (2) in the paper
    if (! (randj <= CR || j == k))
    {
        // <=> vector U(i,j) in the paper
        mutated_individuals[i] = previous_individuals[i];
    }
}

/**
 * Runs on the GPU, called from the CPU or the GPU
*/
__global__ void kernelUpdatePBest(float *positions, float *pBests, float* gBest)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle1[j] = positions[i + j];
        tempParticle2[j] = pBests[i + j];
    }

    if (fitness_function(tempParticle1) < fitness_function(tempParticle2))
    {
        for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
            pBests[i + k] = positions[i + k];
    }
}
__global__ void kernelDEMutation(float *individuals, int *indexMutation, float *mutants, float F) {
    extern __shared__ float sharedMem[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_POPULATION) return;
    
    int r_base = indexMutation[i * 3];
    int r_1 = indexMutation[i * 3 + 1];
    int r_2 = indexMutation[i * 3 + 2];
    
    float *base = &sharedMem[threadIdx.x * NUM_OF_DIMENSIONS * 3];
    float *x_r1 = &base[NUM_OF_DIMENSIONS];
    float *x_r2 = &x_r1[NUM_OF_DIMENSIONS];
    
    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        base[d] = individuals[r_base * NUM_OF_DIMENSIONS + d];
        x_r1[d] = individuals[r_1 * NUM_OF_DIMENSIONS + d];
        x_r2[d] = individuals[r_2 * NUM_OF_DIMENSIONS + d];
    }
    
    __syncthreads();
    
    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        mutants[i * NUM_OF_DIMENSIONS + d] = base[d] + F * (x_r1[d] - x_r2[d]);
    }
}
  
extern "C" void cuda_de(float *population, float* evaluation)
{
    float *devPopulation;
    float *devEval;
    float *devMutants;
    int *devIndexMutation;
    float temp[NUM_OF_DIMENSIONS];
       
    hipMalloc((void**)&devPopulation, sizeof(float) * size);
    hipMalloc((void**)&devEval, sizeof(float) * NUM_OF_POPULATION);
    hipMalloc((void**)&devMutants, sizeof(float) * size);
    hipMalloc((void**)&devIndexMutation, sizeof(int) * NUM_OF_POPULATION * 3);

    int threadsNum = 256;
    int blocksNum = (NUM_OF_POPULATION + threadsNum - 1) / threadsNum;
    int sharedMemSize = threadsNum * NUM_OF_DIMENSIONS * 3 * sizeof(float);
   
    hipMemcpy(devPopulation, population, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devEval, evaluation, sizeof(float) * NUM_OF_POPULATION, hipMemcpyHostToDevice);
  
    for (int iter = 0; iter < MAX_ITER; iter++)
    {    
        kernelInitializePopulation<<<blocksNum, threadsNum>>>(devPopulation);  

        kernelEvaluerPopulation<<<blocksNum, threadsNum>>>(devPopulation, devEval);

        // Génération des indices de mutation (à implémenter)
        // kernelGenerateIndices<<<blocksNum, threadsNum>>>(devIndexMutation);

        kernelDEMutation<<<blocksNum, threadsNum, sharedMemSize>>>(devPopulation, devIndexMutation, devMutants, F);

        int k = getRandomClamped(0, NUM_OF_DIMENSIONS - 1);
        kernelCrossoverDE<<<blocksNum, threadsNum>>>(devPopulation, devMutants, k);
        
        // Ajoutez ici le kernel de sélection si nécessaire
    }

    hipMemcpy(population, devPopulation, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(evaluation, devEval, sizeof(float) * NUM_OF_POPULATION, hipMemcpyDeviceToHost);
   
    hipFree(devPopulation);
    hipFree(devEval);
    hipFree(devMutants);
    hipFree(devIndexMutation);
}
       
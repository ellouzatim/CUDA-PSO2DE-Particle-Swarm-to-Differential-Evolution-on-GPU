#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "kernel.h"

__global__ void kernelDEMutation(float *individuals, int *indexMutation, float *mutants, float F) {
    extern __shared__ float sharedMem[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_PARTICLES) return;
    
    int r_base = indexMutation[i * 3];
    int r_1 = indexMutation[i * 3 + 1];
    int r_2 = indexMutation[i * 3 + 2];
    
    float *base = &sharedMem[threadIdx.x * NUM_OF_DIMENSIONS * 3];
    float *x_r1 = &base[NUM_OF_DIMENSIONS];
    float *x_r2 = &x_r1[NUM_OF_DIMENSIONS];
    
    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        base[d] = individuals[r_base * NUM_OF_DIMENSIONS + d];
        x_r1[d] = individuals[r_1 * NUM_OF_DIMENSIONS + d];
        x_r2[d] = individuals[r_2 * NUM_OF_DIMENSIONS + d];
    }
    
    __syncthreads();
    
    for (int d = 0; d < NUM_OF_DIMENSIONS; d++) {
        mutants[i * NUM_OF_DIMENSIONS + d] = base[d] + F * (x_r1[d] - x_r2[d]);
    }
}